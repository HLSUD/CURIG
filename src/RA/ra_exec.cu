/*
INVERSE: type 1

FORWARD: type 2

*/
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include <hipfft/hipfft.h>

#include "curafft_plan.h"
#include "conv_invoker.h"
#include "deconv_invoker.h"
#include "dft.h"

int exec_inverse(curafft_plan *plan){
    /*
    Currently, just for improved W stacking
    Two different execution flows
        Flow1: the data size is small and memory is sufficent for whole conv
        Flow2: the data size is large, the data is divided into parts 
    */

    if(plan->execute_flow==1){
        /// curafft_conv workflow for enough memory
        checkCudaErrors(hipMemset(plan->fw,0,plan->num_w*plan->nf1*plan->nf2*sizeof(CUCPX)));
        // 1. convlution
        curafft_conv(plan);

        // 2. cufft
        int direction = plan->iflag;
        // cautious, a batch of fft, bath size is num_w when memory is sufficent.
        CUFFT_EXEC(plan->fftplan, plan->fw, plan->fw, direction); // sychronized or not
        // keep the N1*N2*num_w. ignore the outputs that are out of range 

        // 3. dft on w (or 1 dimensional nufft type3)
        curadft_invoker(plan);
        
        // 4. deconvolution (correction)
        curafft_deconv(plan);
        
    }
    else if(plan->execute_flow==2){
        /// curafft_partial_conv workflow for insufficient memory
        
        // offset array with size of 
        for(int i=0; i<plan->num_w; i+=plan->batchsize){
            //memory allocation of fw may cause error, if size is too large, decrease the batchsize.
            checkCudaErrors(hipMemset(plan->fw,0,plan->batchsize*plan->nf1*plan->nf2*sizeof(CUCPX)));
            // 1. convlution
            curafft_conv(plan);

        }
    }
    

}