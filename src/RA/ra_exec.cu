#include "hip/hip_runtime.h"
/*
INVERSE: type 1

FORWARD: type 2

*/
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include <hipfft/hipfft.h>

#include "curafft_plan.h"
#include "conv_invoker.h"
#include "deconv.h"
#include "precomp.h"
#include "ragridder_plan.h"
#include "ra_exec.h"
#include "cuft.h"

__global__ void gridder_rescaling_complex(CUCPX *x, PCS scale_ratio, int N){
    int idx;
    for(idx = blockIdx.x * blockDim.x + threadIdx.x; idx<N; idx += gridDim.x * blockDim.x){
        x[idx].x *= scale_ratio;
        x[idx].y *= scale_ratio;
    }
}

__global__ void div_n_lm(CUCPX *fk, PCS xpixelsize, PCS ypixelsize, int N1, int N2){
    int idx;
    PCS n_lm;
    int row, col;
    for(idx = blockDim.x*blockIdx.x+threadIdx.x; idx<N1*N2; idx+=gridDim.x*blockDim.x){
        row = idx / N1;
        col = idx % N1;
        // printf("%d, %.5lf, %.5lf, %d, %d\n",idx,xpixelsize,ypixelsize,row,col);
        // printf("idx %d, %.4lf\n",idx,sqrt(1 - pow((row-N2/2)*xpixelsize,2) - pow((col-N1/2)*ypixelsize, 2)));
        n_lm = sqrt(1.0 - pow((row-N2/2)*xpixelsize,2) - pow((col-N1/2)*ypixelsize, 2));
        fk[idx].x /= n_lm;
        fk[idx].y /= n_lm;
    }
}

int curaew_scaling(curafft_plan *plan, ragridder_plan *gridder_plan){
    // ending work
    // 1. fourier transform related rescaling
    int N1 = gridder_plan->width;
    int N2 = gridder_plan->height;
    int N = N1*N2;
    //PCS scaling_ratio = 1.0 / gridder_plan->pixelsize_x / gridder_plan->pixelsize_y;
    int blocksize = 256;
    int gridsize = (N-1)/blocksize + 1;
    
    // gridder_rescaling_complex<<<gridsize,blocksize>>>(plan->fk, scaling_ratio, N);
    // checkCudaErrors(hipDeviceSynchronize());
    
    // 2. dividing n_lm
    div_n_lm<<<gridsize,blocksize>>>(plan->fk, gridder_plan->pixelsize_x, gridder_plan->pixelsize_y, N1,N2);
    checkCudaErrors(hipDeviceSynchronize());
    
    return 0;
}

int exec_vis2dirty(curafft_plan *plan, ragridder_plan *gridder_plan)
{
    /*
    Currently, just suitable for improved W stacking
    Two different execution flows
        Flow1: the data size is relatively small and memory is sufficent for whole conv
        Flow2: the data size is too large, the data is divided into parts 
    */
    int ier=0;
    //printf("execute flow %d\n",plan->execute_flow);
    if (plan->execute_flow == 1)
    {
            /// curafft_conv workflow for enough memory
#ifdef DEBUG
            printf("plan info printing...\n");
            printf("nf (%d,%d,%d), upsampfac %lf\n", plan->nf1, plan->nf2, plan->nf3, plan->copts.upsampfac);
            printf("gridder_plan info printing...\n");
            printf("fov %lf, current channel %d, w_s_r %lf\n", gridder_plan->fov, gridder_plan->cur_channel, gridder_plan->w_s_r);
#endif
            // 1. convlution
            ier = curafft_conv(plan);
#ifdef DEBUG
            printf("conv result printing (first w plane)...\n");
            CPX *fw = (CPX *)malloc(sizeof(CPX)*plan->nf1*plan->nf2*plan->nf3);
            hipMemcpy(fw,plan->fw,sizeof(CUCPX)*plan->nf1*plan->nf2*plan->nf3,hipMemcpyDeviceToHost);
            PCS temp =0;
            for(int i=0;i<plan->nf2;i++){
                for(int j=0; j<plan->nf1; j++){
                    temp += fw[i*plan->nf1+j].real();
                    printf("%.3g ",fw[i*plan->nf1+j].real());
                }
                printf("\n");
            }
            printf("fft 000 %.3g\n",temp);
#endif
            // printf("n1 n2 n3 M %d, %d, %d, %d\n",plan->nf1,plan->nf2,plan->nf3,plan->M);
            // 2. cufft
            int direction = plan->iflag;
            // cautious, a batch of fft, bath size is num_w when memory is sufficent.
            CUFFT_EXEC(plan->fftplan, plan->fw, plan->fw, direction); // sychronized or not
            hipDeviceSynchronize();
#ifdef DEBUG
            printf("fft result printing (first w plane)...\n");
            //CPX *fw = (CPX *)malloc(sizeof(CPX)*plan->nf1*plan->nf2*plan->nf3);
            hipMemcpy(fw,plan->fw,sizeof(CUCPX)*plan->nf1*plan->nf2*plan->nf3,hipMemcpyDeviceToHost);
            for(int i=0;i<plan->nf2;i++){
                for(int j=0; j<plan->nf1; j++)
                    printf("%.3g ",fw[i*plan->nf1+j].real());
                printf("\n");
            }
            temp = 0;
            for(int i=0; i<plan->nf3; i++){
                temp += fw[i*plan->nf1*plan->nf2].real();
            }
            printf("dft 00 %.3g\n",temp);
#endif
            // keep the N1*N2*num_w. ignore the outputs that are out of range
            
            // 3. dft on w (or 1 dimensional nufft type3)
            curadft_invoker(plan, gridder_plan->pixelsize_x, gridder_plan->pixelsize_y);
#ifdef DEBUG
            printf("part of dft result printing:...\n");
            //CPX *fw = (CPX *)malloc(sizeof(CPX)*plan->nf1*plan->nf2*plan->nf3);
            hipMemcpy(fw,plan->fw,sizeof(CUCPX)*plan->nf1*plan->nf2*plan->nf3,hipMemcpyDeviceToHost);
            for(int i=0;i<plan->nf2;i++){
                for(int j=0; j<plan->nf1; j++)
                    printf("%.3g ",fw[i*plan->nf1+j].real());
                printf("\n");
            }
#endif
            // 4. deconvolution (correction)
            // error detected, 1. w term deconv
            // 1. 2D deconv towards u and v
            plan->dim = 2;
            ier = curafft_deconv(plan);
#ifdef DEBUG
            printf("deconv result printing stage 1:...\n");
            CPX *fk = (CPX *)malloc(sizeof(CPX)*plan->ms*plan->mt);
            hipMemcpy(fk,plan->fk,sizeof(CUCPX)*plan->ms*plan->mt,hipMemcpyDeviceToHost);
            for(int i=0;i<plan->mt;i++){
                for(int j=0; j<plan->ms; j++)
                    printf("%.5lf ",fk[i*plan->ms+j].real());
                printf("\n");
            }
#endif
            // 2. w term deconv on fk
            ier = curadft_w_deconv(plan, gridder_plan->pixelsize_x, gridder_plan->pixelsize_y);
#ifdef DEBUG
            printf("deconv result printing stage 2:...\n");
            //CPX *fk = (CPX *)malloc(sizeof(CPX)*plan->ms*plan->mt);
            hipMemcpy(fk,plan->fk,sizeof(CUCPX)*plan->ms*plan->mt,hipMemcpyDeviceToHost);
            for(int i=0;i<plan->mt;i++){
                for(int j=0; j<plan->ms; j++)
                    printf("%.5lf ",fk[i*plan->ms+j].real());
                printf("\n");
            }
#endif
            // 5. ending work - scaling
            // /n_lm, fourier related rescale
            curaew_scaling(plan, gridder_plan);
            
    }
    else if (plan->execute_flow == 2)
    {
        /// curafft_partial_conv workflow for insufficient memory

        // offset array with size of
        for (int i = 0; i < gridder_plan->num_w; i += plan->batchsize)
        {
            //memory allocation of fw may cause error, if size is too large, decrease the batchsize.
            checkCudaErrors(hipMemset(plan->fw, 0, plan->batchsize * plan->nf1 * plan->nf2 * sizeof(CUCPX)));
            // 1. convlution
            curafft_conv(plan);
        }
    }
    return ier;
}