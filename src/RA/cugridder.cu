/* --------cugridder-----------
    1. gridder_setting
        fov and other astro related setting
        opt setting
        plan setting
        bin setting
    2. gridder_execution
    3. gridder_destroy
*/

#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include <hipfft/hipfft.h>

#include "conv_invoker.h"
#include "utils.h"


int gridder_setting(int N1, int N2, int method, int kerevalmeth, int w_term_method, double sigma, int iflag,
    int ntransf, int M, PCS *d_u, PCS *d_v, PCS *d_w, CUCPX *d_c, curafft_plan *plan)
{
    /*
        N1, N2 - number of Fouier modes
        method - gridding method
        kerevalmeth - gridding kernel evaluation method
        sigma - upsampling factor
        iflag - flag for fourier transform indicate the direction, HIPFFT_BACKWARD = 1, FORWARD = -1
        ntransf - number of transform
        M - number of nputs (visibility)
        d_u, d_v, d_w - wavelengths in different dimensions
        d_c - value of visibility

        ****issue, degrid
    */
    int ier = 0;
    
    plan = new curafft_plan();
    memset(plan, 0, sizeof(curafft_plan));

    // fov and other astro related setting +++


    // opts and copts setting
    plan->opts.gpu_device_id = 0;
    plan->opts.upsampfac = sigma;
    plan->opts.gpu_sort = 1;
    plan->opts.gpu_binsizex = -1;
    plan->opts.gpu_binsizey = -1;
    plan->opts.gpu_binsizez = -1;
    plan->opts.gpu_kerevalmeth = kerevalmeth;
    plan->opts.gpu_conv_only = 0;
    plan->opts.gpu_gridding_method = method;

    int ier = setup_conv_opts(plan->copts, tol, sigma, kerevalmeth); //check the arguements

	if(ier!=0)printf("setup_error\n");

    // plan setting
    plan->w_term_method = w_term_method;
    // cufft stream malloc in setup_plan
    setup_plan(N1, N2, M, d_u, d_v, d_w, d_c, plan);
    // plan->dim = dim;
	plan->ms = N1;
	plan->mt = N2;
	// plan->mu = nmodes[2];

    int fftsign = (iflag>=0) ? 1 : -1;

	plan->iflag = fftsign;
	plan->ntransf = ntransf;

    if(plan->type == 1)
		plan->copts.direction = 1; //inverse
	if(plan->type == 0)
		plan->copts.direction = 0; //forward

    fwkerhalf1 = (PCS*)malloc(sizeof(PCS)*(nf1/2+1));
    onedim_fseries_kernel(nf1, fwkerhalf1, plan->spopts);//?
    
    fwkerhalf2 = (PCS*)malloc(sizeof(PCS)*(nf2/2+1));
    onedim_fseries_kernel(nf2, fwkerhalf2, plan->spopts);
    
    if(w_term_method){
        // improved_ws
        fwkerhalf3 = (PCS*)malloc(sizeof(PCS)*(nf3/2+1));
        onedim_fseries_kernel(nf3, fwkerhalf3, plan->spopts);
    }

    // copy to device 
    

    checkCudaErrors(hipMemcpy(plan->fwkerhalf1,fwkerhalf1,(nf1/2+1)*
		sizeof(PCS),hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMemcpy(plan->fwkerhalf2,fwkerhalf2,(nf2/2+1)*
		sizeof(PCS),hipMemcpyHostToDevice));
	if(w_term_method)
		checkCudaErrors(hipMemcpy(plan->fwkerhalf3,fwkerhalf3,(nf3/2+1)*
			sizeof(PCS),hipMemcpyHostToDevice));
    

    // cufft plan setting
    hipfftHandle fftplan;
    int n[] = {nf2, nf1};
	int inembed[] = {nf2, nf1};
    // check, multi cufft for different w ??? how to set
	// hipfftCreate(&fftplan);
	// hipfftPlan2d(&fftplan,n[0],n[1],CUFFT_TYPE);
	hipfftPlanMany(&fftplan,2,n,inembed,1,inembed[0]*inembed[1],
		inembed,1,inembed[0]*inembed[1],CUFFT_TYPE,plan->num_w); //need to check and revise
    plan->fftplan = fftplan; 
    

    // set up bin size +++ (for other methods) and related malloc based on gpu method
    // assign memory for index after sorting (can be done in setup_plan)
    // bin sorting (for other methods)
   


    // free host fwkerhalf
    free(fwkerhalf1);
    free(fwkerhalf2);
    if(w_term_method)free(fwkerhalf3);

    return ier;
}


int gridder_exectuion(CUCPX* d_c, CUCPX* d_fk, curafft_plan* plan){

}