#include "hip/hip_runtime.h"
/* --------cugridder-----------
    1. gridder_setting
        fov and other astro related setting
        opt setting
        plan setting
        bin setting
    2. gridder_execution
    3. gridder_destroy
*/

#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include <hipfft/hipfft.h>

#include "conv_invoker.h"
#include "ragridder_plan.h"
#include "curafft_plan.h"
#include "cuft.h"
#include "ra_exec.h"
#include "utils.h"
#include "cugridder.h"


int setup_gridder_plan(int N1, int N2, PCS fov, int lshift, int mshift, int nrow, PCS *d_w, conv_opts copts, ragridder_plan *plan){
    plan->fov = fov;
    plan->width = N1;
    plan->height = N2;
    plan->nrow = nrow;
    // determain number of w 
    // ignore shift
    plan->pixelsize_x = fov / 180.0 * PI / (PCS)N1;
    plan->pixelsize_y = fov / 180.0 * PI / (PCS)N2;
    PCS xpixelsize = plan->pixelsize_x;
    PCS ypixelsize = plan->pixelsize_y;
    PCS l_min = lshift - 0.5*xpixelsize * N1;
    PCS l_max = l_min + xpixelsize * (N1-1);
    
    PCS m_min = mshift - 0.5*ypixelsize * N2;
    PCS m_max = m_min + ypixelsize * (N2-1);

    double upsampling_fac = copts.upsampfac;
    PCS n_lm = sqrt(1 - pow(l_max,2) + pow(m_max, 2));
    // nshift = (no_nshift||(!do_wgridding)) ? 0. : -0.5*(nm1max+nm1min);
    PCS max, min;
    PCS delta_w = 1/(2*upsampling_fac*abs(n_lm-1));
    plan->delta_w = delta_w;
    get_max_min(max, min, d_w, plan->nrow);

    plan->w_max = max;
    plan->w_min = min;
    PCS w_0 = plan->w_min - delta_w * (copts.kw - 1); // first plane
    plan->w_0 = w_0;
    plan->num_w = (plan->w_max - plan->w_min)/delta_w + copts.kw; // another plan

    return 0;
}

// the bin sort should be completed at gridder_settting

int gridder_setting(int N1, int N2, int method, int kerevalmeth, int w_term_method, PCS tol, int direction, double sigma, int iflag,
    int batchsize, int M, int channel, PCS fov, visibility *pointer_v, PCS *d_u, PCS *d_v, PCS *d_w,
    CUCPX *d_c, curafft_plan *plan, ragridder_plan *gridder_plan)
{
    /*
        N1, N2 - number of Fouier modes
        method - gridding method
        kerevalmeth - gridding kernel evaluation method
        tol - tolerance (epsilon)
        direction - 1 HIPFFT_BACKWARD, 0 HIPFFT_FORWARD
        sigma - upsampling factor
        iflag - flag for fourier transform
        batchsize - number of batch in  cufft (used for handling piece by piece)
        M - number of nputs (visibility)
        channel - number of channels
        wgt - weight
        freq - frequency
        d_u, d_v, d_w - wavelengths in different dimensions, x is on host, d_x is on device
        d_c - value of visibility

        ****issue, degridding
    */
    int ier = 0;
    
    plan = new curafft_plan();
    gridder_plan = new ragridder_plan();
    memset(plan, 0, sizeof(curafft_plan));
    memset(gridder_plan, 0, sizeof(ragridder_plan));

    // fov and other astro related setting +++


    // opts and copts setting
    plan->opts.gpu_device_id = 0;
    plan->opts.upsampfac = sigma;
    plan->opts.gpu_sort = 1;
    plan->opts.gpu_binsizex = -1;
    plan->opts.gpu_binsizey = -1;
    plan->opts.gpu_binsizez = -1;
    plan->opts.gpu_kerevalmeth = kerevalmeth;
    plan->opts.gpu_conv_only = 0;
    plan->opts.gpu_gridder_method = method;

    ier = setup_conv_opts(plan->copts, tol, sigma, 0, direction, kerevalmeth); //check the arguements

	if(ier!=0)printf("setup_error\n");

    // plan setting
    // cuda stream malloc in setup_plan
    gridder_plan->channel = channel;
    gridder_plan->w_term_method = w_term_method;
    gridder_plan->speedoflight = SPEEDOFLIGHT;
    gridder_plan->kv.u = pointer_v->u;
    gridder_plan->kv.v = pointer_v->v;
    gridder_plan->kv.w = pointer_v->w;
    gridder_plan->kv.vis = pointer_v->vis;
    gridder_plan->kv.weight = pointer_v->weight;
    gridder_plan->kv.frequency = pointer_v->frequency;
    gridder_plan->kv.pirange = pointer_v->pirange;
    setup_gridder_plan(N1,N2,fov,0,0,M,d_w,plan->copts,gridder_plan);
    

    int nf1 = get_num_cells(N1,plan->copts);
    int nf2 = get_num_cells(N2,plan->copts);
    int nf3 = gridder_plan->num_w;
    
    if(w_term_method)plan->dim =3;
    else plan->dim = 2;
    setup_plan(nf1, nf2, nf3, M, d_u, d_v, d_w, d_c, plan);
    
    // plan->dim = dim;
	plan->ms = N1;
	plan->mt = N2;
	// plan->mu = nmodes[2];

    int fftsign = (iflag>=0) ? 1 : -1;

	plan->iflag = fftsign;
    if (batchsize == 0) batchsize = min(4,gridder_plan->num_w);
	plan->batchsize = batchsize;

    plan->copts.direction = direction; // 1 inverse, 0 forward

    // // fw allocation
    // checkCudaErrors(hipMalloc((void**)&plan->fw,sizeof(CUCPX)*nf1*nf2*nf3));

    PCS *fwkerhalf1 = (PCS*)malloc(sizeof(PCS)*(plan->nf1/2+1));
    onedim_fseries_kernel(plan->nf1, fwkerhalf1, plan->copts); // used for correction
    
    PCS *fwkerhalf2 = (PCS*)malloc(sizeof(PCS)*(plan->nf2/2+1));
    onedim_fseries_kernel(plan->nf2, fwkerhalf2, plan->copts);

    // copy to device 
    checkCudaErrors(hipMemcpy(plan->fwkerhalf1,fwkerhalf1,(plan->nf1/2+1)*
		sizeof(PCS),hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMemcpy(plan->fwkerhalf2,fwkerhalf2,(plan->nf2/2+1)*
		sizeof(PCS),hipMemcpyHostToDevice));
    PCS *fwkerhalf3;
	if(w_term_method){
		// improved_ws
        fwkerhalf3 = (PCS*)malloc(sizeof(PCS)*(plan->nf3/2+1));
        //need to revise
        onedim_fseries_kernel(gridder_plan->num_w, fwkerhalf3, plan->copts);
        checkCudaErrors(hipMemcpy(plan->fwkerhalf3,fwkerhalf3,(gridder_plan->num_w/2+1)*
			sizeof(PCS),hipMemcpyHostToDevice));
    }
    

    // cufft plan setting
    hipfftHandle fftplan;
    int n[] = {N2, N1};
    int inembed[] = {plan->nf2, plan->nf1};
	int onembed[] = {N2, N1};
    
    // check, multi cufft for different w ??? how to set
	// hipfftCreate(&fftplan);
	// hipfftPlan2d(&fftplan,n[0],n[1],CUFFT_TYPE);
    // the bach size sets as the num of w when memory is sufficent. Alternative way, set as a smaller number when memory is insufficient.
    // and handle this piece by piece 
	hipfftPlanMany(&fftplan,2,n,inembed,1,inembed[0]*inembed[1],
		onembed,1,onembed[0]*onembed[1],CUFFT_TYPE,plan->nf3); //need to check and revise (the partial conv will be differnt)
    plan->fftplan = fftplan; 

    // set up bin size +++ (for other methods) and related malloc based on gpu method
    // assign memory for index after sorting (can be done in setup_plan)
    // bin sorting (for other methods)

    // free host fwkerhalf
    free(fwkerhalf1);
    free(fwkerhalf2);
    if(w_term_method)free(fwkerhalf3);

    return ier;
}


int gridder_exectuion(curafft_plan* plan, ragridder_plan* gridder_plan){
    /*
    Execute conv, fft, dft, correction for different direction (gridding or degridding)
    */
    int ier=0;
    // Mult-GPU support: set the CUDA Device ID:
        // int orig_gpu_device_id;
        // hipGetDevice(& orig_gpu_device_id);
        // hipSetDevice(d_plan->opts.gpu_device_id);

	int direction = plan->copts.direction;
    if (direction == 1){
        ier = exec_inverse(plan, gridder_plan);
    }
    else{
        // forward not implement yet
        ier = 0;
    }
	

    // Multi-GPU support: reset the device ID
    // hipSetDevice(orig_gpu_device_id);
    return ier;
}

int gridder_destroy(curafft_plan *plan, ragridder_plan *gridder_plan){
    // free memory
    int ier=0;
    curafft_free(plan);
    free(plan);
    free(gridder_plan->dirty_image);
    free(gridder_plan->kv.u);
    free(gridder_plan->kv.v);
    free(gridder_plan->kv.w);
    free(gridder_plan->kv.vis);
    free(gridder_plan->kv.frequency);
    free(gridder_plan->kv.weight);
    // free(gridder_plan->kv.flag);
    free(gridder_plan);
    return ier;
}