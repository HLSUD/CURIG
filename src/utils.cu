#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <thrust/device_ptr.h>
#include "dataType.h"

void prefix_scan(PCS *d_arr, PCS *d_res, int n, int flag){
    /*
        n - number of elements
        flag - 1 inclusive, 0 exclusive
        Will the output at d_res
        thrust::inclusive_scan(d_arr, d_arr + n, d_res);???
    */
    thrust::device_ptr<PCS> d_ptr(d_arr); // not convert
	thrust::device_ptr<PCS> d_result(d_res);
    if(flag)
    thrust::inclusive_scan(d_ptr, d_ptr + n, d_result);
    else
	thrust::exclusive_scan(d_ptr, d_ptr + n, d_result);
}

void get_max_min(PCS &max, PCS &min, PCS *d_array, int n){
    /*
        Get the maximum and minimum values of array by thrust
        Will be fast with one invokation getting max and min?
    */
    thrust::device_ptr<PCS> d_ptr = thrust::device_pointer_cast(d_array);
    PCS *temp = thrust::max_element(d_ptr, d_ptr+n);
    max = *temp;
    temp = thrust::min_element(d_ptr, d_ptr+n);
    min = *temp;
}

void GPU_info(){
    /*
    int *h_max_test, *h_max_test2, *h_max_test3;
    CHECK(hipMalloc(&h_max_test,sizeof(float)*1024*1024));
    CHECK(
    hipMalloc(&h_max_test2,sizeof(float)*1024*1024));
    CHECK(
    hipMalloc(&h_max_test3,sizeof(float)*1024*1024*1000));
    hipFree(h_max_test);
    */
    printf("%s Starting... \n");
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess){
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        exit(EXIT_FAILURE);
    }

    if (deviceCount==0) printf("There is no available device that support CUDA\n");
    else printf("Detected %d CUDA capable device(s)\n",deviceCount);

    int dev, driverVersion = 0, runtimeVersion = 0;

    dev = 0;

    printf("Input the device index:");
    scanf("%d",&dev);
    hipSetDevice(dev);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,dev);
    printf("Device %d: %s\n",dev,deviceProp.name);

    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf("CUDA Driver Version / Runtime Version       %d.%d / %d.%d\n",
    driverVersion/1000,driverVersion%1000, runtimeVersion/1000, runtimeVersion%1000);
    printf("CUDA Capability Major/Minor version number: %d.%d\n", deviceProp.major, deviceProp.minor);
    printf("Total amount of global memory:              %.2f MBytes\n",(float)deviceProp.totalGlobalMem/(pow(1024.0,2)));
    printf("GPU clock rate:                             %.0f MHz\n",deviceProp.clockRate * 1e-3f);
    printf("Memory clock rate:                          %.0f MHz\n",deviceProp.memoryClockRate * 1e-3f);
    printf("Memory Bus Width:                           %d-bit\n", deviceProp.memoryBusWidth);

    if(deviceProp.l2CacheSize){
        printf("L2 Cache Size:                          %d bytes\n",deviceProp.l2CacheSize);
    }
    printf("Total amount of constant memory:            %lu bytes\n", deviceProp.totalConstMem);
    printf("Total amount of shared memory per block:    %lu bytes\n", deviceProp.sharedMemPerBlock);
    printf("Total number of registers available per block: %d\n",deviceProp.regsPerBlock);
    printf("Warp size:                                  %d\n",deviceProp.warpSize);
    printf("Number of multiprocessors:                  %d\n", deviceProp.multiProcessorCount);
    printf("Maximum number of threads per multiprocessor: %d\n",deviceProp.maxThreadsPerMultiProcessor);
    //printf("Maximum number of blocks per multiprocessor: %d\n",deviceProp.maxBlocksPerMultiProcessor);
    printf("Maximum number of thread per block:          %d\n",deviceProp.maxThreadsPerBlock);
    printf("Maximum sizes of each dimension of a block: %d x %d x %d\n",deviceProp.maxThreadsDim[0],deviceProp.maxThreadsDim[1]
    ,deviceProp.maxThreadsDim[2]);
    printf("Maximum sizes of each dimension of a grid:  %d x %d x %d\n",deviceProp.maxGridSize[0],deviceProp.maxGridSize[1],
    deviceProp.maxGridSize[2]);
    printf("Maximum memory pitch:                       %lu bytes\n",deviceProp.memPitch);

}