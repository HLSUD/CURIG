#include "hip/hip_runtime.h"
/*
Invoke conv related kernel
*/

//#include "../memtransfer.h"
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>
#include <assert.h>
#include <hip/hip_complex.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include "dataType.h"
#include "conv_invoker.h"
#include "conv.h"

/*
int setup_conv_opts(conv_opts *c_opts, PCS eps, PCS upsampfac, int kerevalmeth){
  int ier = 0;
  kerevalmeth = 1;
  return ier;
}
*/

/*
void get_max_min(PCS *x, int num, PCS *h_res){
  PCS *d_res;
  CHECK(hipMalloc((void **)&d_res,sizeof(PCS)*2));
  reduce_max_min<<<(num-1)/BLOCKSIZE+1,BLOCKSIZE>>>(x,num,d_res);
  CHECK(cudaDeviceS...);
  CHECK(hipMemcpy(h_res,d_res,sizeof(PCS)*2),hipMemcpyDeviceToHost);
  hipFree(d_res);
}
*/

int setup_plan(int nf1, int nf2, int M, PCS *d_u, PCS *d_v, PCS *d_w, CUCPX *d_c, curafft_plan *plan)
{
  /* different dim will have different setting
    ----plan setting, and related memory allocation----
        nf1, nf2 - number of UPTS (resolution of image)
        M - number of NUPTS (num of vis)
        d_u, d_v, d_w - location
        d_c - value
    */
  int ier = 0;
  //wrong here
  /*
  plan->kv.u = d_u;
  plan->kv.v = d_v;
  plan->kv.w = d_w;
  plan->kv.vis = d_c;
  */ 
  //int ier;
  plan->nf1 = nf1;
  plan->nf2 = nf2;

  /*
  //get number of w
  int num_w = 0;
  //reduce to get maximum and minimum, h_res[0] max, [1] min
  PCS *h_res = (PCS *)malloc(sizeof(int)*2);
  PCS max = h_res[0];
  PCS min = h_res[1];
  free(h_res);
  PCS n_scale = sqrt(max(1. - l_max * l_max - m_max * m_max, 0.)) - 1.;
  if (l_max * l_max + m_max * m_max > 1.)
    n_scale = -sqrt(abs(1. - l_max * l_max - m_max * m_max)) - 1.;
  plan->num_w =  abs(n_scale)/(0.25) * (max-min) + plan->copts.kw;
  */
  plan->num_w = 2 * nf1;

  plan->M = M;
  //plan->maxbatchsize = 1;

  plan->byte_now = 0;
  // No extra memory is needed in nuptsdriven method (case 1)
  switch (plan->opts.gpu_method)
  {
  case 0:
  {
    if (plan->opts.gpu_sort)
    {
      CHECK(hipMalloc(&plan->cell_loc, sizeof(INT_M) * M));
    }
  }
  case 1:
  {
    if (plan->opts.gpu_sort)
    {
      CHECK(hipMalloc(&plan->cell_loc, sizeof(INT_M) * M));
    }
  }
  break;
  
  default:
    std::cerr << "err: invalid method " << std::endl;
  }
  return ier;
}

  int ws_conv(int nf1, int nf2, int nf3, int M, curafft_plan *plan)
  {
    return 0;
  }

  int improved_ws_conv(int nf1, int nf2, int nf3, int M, curafft_plan *plan)
  {
    //add content

    dim3 grid;
    dim3 block;
    if (plan->opts.gpu_method == 0)
    {
      block.x = 256;
      grid.x = (M - 1) / block.x + 1;
      conv_3d_nputsdriven<<<grid, block>>>(plan->kv.u, plan->kv.v, plan->kv.w, plan->kv.vis, plan->fw, plan->M,
                                           plan->copts.kw, nf1, nf2, nf3, plan->copts.ES_c, plan->copts.ES_beta, plan->copts.pirange, plan->cell_loc);
    }

    return 0;
  }

  int curafft_conv(curafft_plan * plan)
  {
    /*
    ---- convolution opertion ----
    */

    int ier = 0;
    int nf1 = plan->nf1;
    int nf2 = plan->nf2;
    int nf3 = plan->num_w;
    int M = plan->M;
    if (plan->w_term_method == 0)
    {
      ws_conv(nf1, nf2, nf3, M, plan);
    }
    if (plan->w_term_method == 1)
    {
      //get nupts location in grid cells
      improved_ws_conv(nf1, nf2, nf3, M, plan);
    }
    return ier;
  }
