#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <thrust/complex.h>
#include <algorithm>
//#include <thrust>
using namespace thrust;


#include "ragridder_plan.h"
#include "conv_invoker.h"
#include "cuft.h"
#include "deconv.h"
#include "cugridder.h"
#include "precomp.h"
#include "utils.h"


int main(int argc, char *argv[])
{
	/* Input: M, N1, N2, epsilon method
		method - conv method
		M - number of randomly distributed points
		N1, N2 - output size
		epsilon - tolerance
	*/
	int ier = 0;
	int N = 16;
	PCS sigma = 2.0; // upsampling factor
	int M = 30;

	
	PCS epsilon = 1e-6;
	
	int kerevalmeth = 0;
	
	int method=0;

	//gpu_method == 0, nupts driven

	//int ier;
	PCS *u;
	CPX *c;
	u = (PCS *)malloc(M  * sizeof(PCS)); //Allocates page-locked memory on the host.
	c = (CPX *)malloc(M  * sizeof(CPX));
	PCS *d_u;
	CUCPX *d_c, *d_fk;
	CUCPX *d_fw;
	checkCudaErrors(hipMalloc(&d_u, M  * sizeof(PCS)));
	checkCudaErrors(hipMalloc(&d_c, M  * sizeof(CUCPX)));
    /// pixel size 
	// generating data
	for (int i = 0; i < M; i++)
	{
		u[i] = randm11()*PI; //xxxxx
		c[i].real(randm11()); // M vis per channel, weight?
		c[i].imag(randm11());
		// wgt[i] = 1;
	}

	PCS *k = (PCS*) malloc(sizeof(PCS)*N*10);
	// PCS pixelsize = 0.01;
	for (size_t i = 0; i < N; i++)
	{
		/* code */
		k[i] = (int)i-N/2;
		// k[i] = randm11();
	}
	
	// double a[5] = {-PI/2, -PI/3, 0, PI/3, PI/2}; // change to random data
	// for(int i=0; i<M; i++){
	// 	u[i] = a[i/5];
	// 	v[i] = a[i%5];
	// }
#ifdef DEBUG
	printf("origial input data...\n");
	for(int i=0; i<M; i++){
		printf("%.3lf ",u[i]);
	}
	printf("\n");
	for(int i=0; i<M; i++){
		printf("%.3lf ",c[i].real());
	}
	printf("\n");
#endif
	// ignore the tdirty
	// how to convert ms to vis

	//printf("generated data, x[1] %2.2g, y[1] %2.2g , z[1] %2.2g, c[1] %2.2g\n",x[1] , y[1], z[1], c[1].real());

	// Timing begin
	//data transfer
	checkCudaErrors(hipMemcpy(d_u, u, M * sizeof(PCS), hipMemcpyHostToDevice)); //u
	checkCudaErrors(hipMemcpy(d_c, c, M * sizeof(CUCPX), hipMemcpyHostToDevice));

	/* ----------Step2: plan setting------------*/
	curafft_plan *plan;

	plan = new curafft_plan();
    memset(plan, 0, sizeof(curafft_plan));

	int direction = 1; //inverse
	
	// opts and copts setting
    plan->opts.gpu_device_id = 0;
    plan->opts.upsampfac = sigma;
    plan->opts.gpu_sort = 1;
    plan->opts.gpu_binsizex = -1;
    plan->opts.gpu_binsizey = -1;
    plan->opts.gpu_binsizez = -1;
    plan->opts.gpu_kerevalmeth = kerevalmeth;
    plan->opts.gpu_conv_only = 0;
    plan->opts.gpu_gridder_method = method;

    ier = setup_conv_opts(plan->copts, epsilon, sigma, 1, direction, kerevalmeth); //check the arguements

	if(ier!=0)printf("setup_error\n");

    // plan setting
    // cuda stream malloc in setup_plan
    

    int nf1 = get_num_cells(M,plan->copts);

    
    plan->dim = 1;
    setup_plan(nf1, 1, 1, M, d_u, NULL, NULL, d_c, plan);

	plan->ms = N; ///!!!
	plan->mt = 1;
	plan->mu = 1;
    plan->execute_flow = 1;
	int iflag = direction;
    int fftsign = (iflag>=0) ? 1 : -1;

	plan->iflag = fftsign; //may be useless| conflict with direction
	plan->batchsize = 1;

    plan->copts.direction = direction; // 1 inverse, 0 forward
    PCS *d_fwkerhalf;
    checkCudaErrors(hipMalloc((void**)&d_fwkerhalf,sizeof(PCS)*(N)));
	hipMemset(d_fwkerhalf,0,sizeof(PCS)*N);
    PCS *d_k;
    checkCudaErrors(hipMalloc((void**)&d_k,sizeof(PCS)*(N)));
    checkCudaErrors(hipMemcpy(d_k,k,sizeof(PCS)*(N),hipMemcpyHostToDevice));
    fourier_series_appro_invoker(d_fwkerhalf,d_k,plan->copts, N,nf1/2+1); // correction with k, may be wrong, k will be free in this function
	
	
	// printf("begining...\n");
	// fourier_series_appro_invoker(d_fwkerhalf,plan->copts,nf1/2+1);
	PCS *fwkerhalf = (PCS *)malloc(sizeof(PCS)*(N));
	hipMemcpy(fwkerhalf, d_fwkerhalf, sizeof(PCS)*(N), hipMemcpyDeviceToHost);
	printf("correction factor printing method1...\n");
	for (size_t i = 0; i < N; i++)
	{
		/* code */
		printf("%lf ",fwkerhalf[i]);
	}
	printf("\n");
	fourier_series_appro_invoker(d_fwkerhalf,plan->copts,nf1/2+1);
	hipMemcpy(fwkerhalf, d_fwkerhalf, sizeof(PCS)*(nf1/2+1), hipMemcpyDeviceToHost);
	printf("correction factor printing method2...\n");
	for (size_t i = 0; i < N/2+1; i++)
	{
		/* code */
		printf("%lf ",fwkerhalf[i]);
	}
	printf("\n");
	// for(int i=0; i<nf1/2+1; i++){
	// 	k[i] = i;
	// }
	// PCS *d_temp;
	// checkCudaErrors(hipMalloc((void**)&d_temp,sizeof(PCS)*(nf1/2+1)));
	// checkCudaErrors(hipMemcpy(d_temp,k,sizeof(PCS)*(nf1/2+1),hipMemcpyHostToDevice));
	// fourier_series_appro_invoker(d_fwkerhalf,d_temp,plan->copts,nf1/2+1,nf1/2+1);
	// hipMemcpy(fwkerhalf, d_fwkerhalf, sizeof(PCS)*(N), hipMemcpyDeviceToHost);
	// printf("correction factor printing method2...\n");
	// for (size_t i = 0; i < N; i++)
	// {
	// 	/* code */
	// 	printf("%lf ",fwkerhalf[i]);
	// }
	// printf("\n");

    
	// fw (conv res set)
	checkCudaErrors(hipMalloc((void**)&d_fw,sizeof(CUCPX)*nf1));
	checkCudaErrors(hipMemset(d_fw, 0, sizeof(CUCPX)*nf1));
	plan->fw = d_fw;
	// fk malloc and set
	checkCudaErrors(hipMalloc((void**)&d_fk,sizeof(CUCPX)*N));
	plan->fk = d_fk;

	// calulating result
	curafft_conv(plan);
	CPX *fw = (CPX *)malloc(sizeof(CPX)*nf1);
	hipMemcpy(fw,plan->fw,sizeof(CUCPX)*nf1,hipMemcpyDeviceToHost);
#ifdef DEBUG
	printf("conv result printing...\n");
	
	PCS temp_res=0;
	
	for (size_t i = 0; i < nf1; i++)
	{
		/* code */
		printf("%lf ",fw[i].real());
	}
	printf("\n");
	
#endif
	CPX *fk = (CPX *)malloc(sizeof(CPX)*N);
	memset(fk,0,sizeof(CPX)*N);
	// dft
	for (size_t i = 0; i < N; i++)
	{
		/* code */
		for (size_t j = 0; j < nf1; j++)
		{

			fk[i] += fw[j<nf1/2?j+nf1/2:j-nf1/2]*exp(k[i]*(j)/(PCS)nf1*2*PI*IMA);
		}
		
	}
	
printf("dft result printing...\n");
for (size_t i = 0; i < N; i++)
{
	/* code */
	printf("%lf ",fk[i].real());
}
printf("\n");

	 hipfftHandle fftplan;
    int n[] = { plan->nf1};
    int inembed[] = { plan->nf1};
	int onembed[] = { plan->nf1};
    
	// hipfftCreate(&fftplan);
	// hipfftPlan2d(&fftplan,n[0],n[1],CUFFT_TYPE);
    // the bach size sets as the num of w when memory is sufficent. Alternative way, set as a smaller number when memory is insufficient.
    // and handle this piece by piece 
    
	hipfftPlanMany(&fftplan,1,n,inembed,1,inembed[0],
		onembed,1,onembed[0],CUFFT_TYPE,plan->nf3); //need to check and revise (the partial conv will be differnt)
    plan->fftplan = fftplan; 

	// printf("correction factor printing...\n");
	// for(int i=0; i<N1/2; i++){
	// 	printf("%.3g ",fwkerhalf1[i]);
	// }
	// printf("\n");
	// for(int i=0; i<N2/2; i++){
	// 	printf("%.3g ",fwkerhalf2[i]);
	// }
	// printf("\n");
	// deconv
	//PCS *fwkerhalf = (PCS *)malloc(sizeof(PCS)*(N));
	hipMemcpy(fwkerhalf, d_fwkerhalf, sizeof(PCS)*(N), hipMemcpyDeviceToHost);

	for(int i=0; i<N; i++){
		fk[i] = fk[i] / fwkerhalf[abs(i-N/2)];
	}

	
	// result printing
	printf("final result printing...\n");
	for(int i=0; i<N; i++){
		printf("%.10lf ",fk[i].real());
		
	}
	printf("\n");
	printf("ground truth printing...\n");
	for (size_t i = 0; i < N; i++)
	{
		/* code */
		fk[i] = 0;
		for(int j=0; j<M; j++){
			fk[i] += c[j]*exp(k[i]*u[j]*IMA);
		}
	}
	
	for(int i=0; i<N; i++){
		printf("%.10lf ",fk[i].real());
		
	}
	printf("\n");
	
	//free
	curafft_free(plan);
	free(fk);
	free(u);
	free(c);

	return ier;
}