#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <thrust/complex.h>
#include <algorithm>
//#include <thrust>
using namespace thrust;


#include "ragridder_plan.h"
#include "conv_invoker.h"
#include "cuft.h"
#include "deconv.h"
#include "cugridder.h"
#include "precomp.h"
#include "utils.h"


int main(int argc, char *argv[])
{
	/* Input: M, N1, N2, epsilon method
		method - conv method
		M - number of randomly distributed points
		N1, N2 - output size
		epsilon - tolerance
	*/
	int ier = 0;
	int N = 16;
	PCS sigma = 2.0; // upsampling factor // for not on grid points needs larger upsampling factor
	int M = 30;

	
	PCS epsilon = 1e-6;
	
	int kerevalmeth = 0;
	
	int method=0;

	//gpu_method == 0, nupts driven

	//int ier;
	PCS *u;
	CPX *c;
	u = (PCS *)malloc(M  * sizeof(PCS)); //Allocates page-locked memory on the host.
	c = (CPX *)malloc(M  * sizeof(CPX));
	PCS *d_u;
	CUCPX *d_c, *d_fk;
	CUCPX *d_fw;
	checkCudaErrors(hipMalloc(&d_u, M  * sizeof(PCS)));
	checkCudaErrors(hipMalloc(&d_c, M  * sizeof(CUCPX)));
    /// pixel size 
	// generating data
	for (int i = 0; i < M; i++)
	{
		u[i] = randm11()*PI; //xxxxx
		c[i].real(randm11()); // M vis per channel, weight?
		c[i].imag(randm11());
		// wgt[i] = 1;
	}

	PCS *k = (PCS*) malloc(sizeof(PCS)*N*10);
	// PCS pixelsize = 0.01;
	for (size_t i = 0; i < N; i++)
	{
		/* code */
		 k[i] = (int)i-N/2;
		// k[i] = -abs(randm11());
		// k[i] = i/(double)N;
	}
	
	
	//data transfer
	checkCudaErrors(hipMemcpy(d_u, u, M * sizeof(PCS), hipMemcpyHostToDevice)); //u
	checkCudaErrors(hipMemcpy(d_c, c, M * sizeof(CUCPX), hipMemcpyHostToDevice));

	/* ----------Step2: plan setting------------*/
	curafft_plan *plan;

	plan = new curafft_plan();
    memset(plan, 0, sizeof(curafft_plan));

	int direction = 1; //inverse
	
	// opts and copts setting
    plan->opts.gpu_device_id = 0;
    plan->opts.upsampfac = sigma;
    plan->opts.gpu_sort = 1;
    plan->opts.gpu_binsizex = -1;
    plan->opts.gpu_binsizey = -1;
    plan->opts.gpu_binsizez = -1;
    plan->opts.gpu_kerevalmeth = kerevalmeth;
    plan->opts.gpu_conv_only = 0;
    plan->opts.gpu_gridder_method = method;

    ier = setup_conv_opts(plan->copts, epsilon, sigma, 1, direction, kerevalmeth); //check the arguements

	if(ier!=0)printf("setup_error\n");

    // plan setting
    // cuda stream malloc in setup_plan
    

    int nf1 = get_num_cells(M,plan->copts);
	//printf("nf: %d\n",nf1);
    //printf("copt info kw %d, upsampfac %lf, beta %lf\n",plan->copts.kw,plan->copts.upsampfac,plan->copts.ES_beta);
    plan->dim = 1;
    setup_plan(nf1, 1, 1, M, d_u, NULL, NULL, d_c, plan);

	plan->ms = N; ///!!!
	plan->mt = 1;
	plan->mu = 1;
    plan->execute_flow = 1;
	int iflag = direction;
    int fftsign = (iflag>=0) ? 1 : -1;

	plan->iflag = fftsign; //may be useless| conflict with direction
	plan->batchsize = 1;

    plan->copts.direction = direction; // 1 inverse, 0 forward
    PCS *d_fwkerhalf;
    checkCudaErrors(hipMalloc((void**)&d_fwkerhalf,sizeof(PCS)*(N)));
	hipMemset(d_fwkerhalf,0,sizeof(PCS)*N);
    PCS *d_k;
    checkCudaErrors(hipMalloc((void**)&d_k,sizeof(PCS)*(N)));
    checkCudaErrors(hipMemcpy(d_k,k,sizeof(PCS)*(N),hipMemcpyHostToDevice));
    fourier_series_appro_invoker(d_fwkerhalf,d_k,plan->copts, N,nf1/2+1); // correction with k, may be wrong, k will be free in this function
	
	
	// printf("begining...\n");
	// fourier_series_appro_invoker(d_fwkerhalf,plan->copts,nf1/2+1);
	PCS *fwkerhalf = (PCS *)malloc(sizeof(PCS)*(N));
	hipMemcpy(fwkerhalf, d_fwkerhalf, sizeof(PCS)*(N), hipMemcpyDeviceToHost);
#ifdef DEBUG
	printf("correction factor printing method1...\n");
	for (size_t i = 0; i < N; i++)
	{
		/* code */
		printf("%lf ",fwkerhalf[i]);
	}
	printf("\n");
#endif
	// fw (conv res set)
	checkCudaErrors(hipMalloc((void**)&d_fw,sizeof(CUCPX)*nf1));
	checkCudaErrors(hipMemset(d_fw, 0, sizeof(CUCPX)*nf1));
	plan->fw = d_fw;
	// fk malloc and set
	checkCudaErrors(hipMalloc((void**)&d_fk,sizeof(CUCPX)*N));
	plan->fk = d_fk;

	// calulating result
	curafft_conv(plan);
	CPX *fw = (CPX *)malloc(sizeof(CPX)*nf1);
	hipMemcpy(fw,plan->fw,sizeof(CUCPX)*nf1,hipMemcpyDeviceToHost);
#ifdef DEBUG
	printf("conv result printing...\n");
	
	for (size_t i = 0; i < nf1; i++)
	{
		/* code */
		printf("%lf ",fw[i].real());
	}
	printf("\n");
	
#endif
	CPX *fk = (CPX *)malloc(sizeof(CPX)*N);
	memset(fk,0,sizeof(CPX)*N);
	// dft
	for (size_t i = 0; i < N; i++)
	{
		/* code */
		for (size_t j = 0; j < nf1; j++)
		{
			if(j<nf1/2){
                fk[i] += fw[j+nf1/2]*exp(k[i]*((j)/((PCS)nf1)*2.0*PI*IMA));
            }
            else{
                fk[i] += fw[j-nf1/2]*exp(k[i]*( (j-(PCS)nf1)/((PCS)nf1) )*2.0*PI*IMA); //fw[j-nf1/2]*exp(k[i]*( (j-nf1)/((PCS)nf1) )*2.0*PI*IMA); not work why
            }
		}
		
	}
#ifdef DEBUG
	printf("dft result printing...\n");
	for (size_t i = 0; i < N; i++)
	{
		/* code */
		printf("%lf ",fk[i].real());
	}
	printf("\n");
#endif
	

	// printf("correction factor printing...\n");
	// for(int i=0; i<N1/2; i++){
	// 	printf("%.3g ",fwkerhalf1[i]);
	// }
	// printf("\n");
	// for(int i=0; i<N2/2; i++){
	// 	printf("%.3g ",fwkerhalf2[i]);
	// }
	// printf("\n");
	// deconv
	//PCS *fwkerhalf = (PCS *)malloc(sizeof(PCS)*(N));
	//hipMemcpy(fwkerhalf, d_fwkerhalf, sizeof(PCS)*(N), hipMemcpyDeviceToHost);

	for(int i=0; i<N; i++){
		fk[i] = fk[i] / fwkerhalf[i];
	}

	
	// result printing
	printf("final result printing...\n");
	for(int i=0; i<N; i++){
		printf("%.10lf ",fk[i].real());
		
	}
	printf("\n");
	printf("ground truth printing...\n");
	for (size_t i = 0; i < N; i++)
	{
		/* code */
		fk[i] = 0;
		for(int j=0; j<M; j++){
			fk[i] += c[j]*exp(k[i]*u[j]*IMA);
		}
	}
	
	for(int i=0; i<N; i++){
		printf("%.10lf ",fk[i].real());
		
	}
	printf("\n");
	
	//free
	curafft_free(plan);
	free(fk);
	free(u);
	free(c);

	return ier;
}